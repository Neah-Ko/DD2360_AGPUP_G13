

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

#define BLOCK_SIZE  16
#define HEADER_SIZE 138

#define BLOCK_SIZE_SH 18

typedef unsigned char BYTE;

/**
 * Structure that represents a BMP image.
 */
typedef struct
{
    int   width;
    int   height;
    float *data;
} BMPImage;

typedef struct timeval tval;

BYTE g_info[HEADER_SIZE]; // Reference header

/**
 * Reads a BMP 24bpp file and returns a BMPImage structure.
 * Thanks to https://stackoverflow.com/a/9296467
 */
BMPImage readBMP(char *filename)
{
    BMPImage bitmap = { 0 };
    int      size   = 0;
    BYTE     *data  = NULL;
    FILE     *file  = fopen(filename, "rb");
    
    // Read the header (expected BGR - 24bpp)
    fread(g_info, sizeof(BYTE), HEADER_SIZE, file);

    // Get the image width / height from the header
    bitmap.width  = *((int *)&g_info[18]);
    bitmap.height = *((int *)&g_info[22]);
    size          = *((int *)&g_info[34]);
    
    // Read the image data
    data = (BYTE *)malloc(sizeof(BYTE) * size);
    fread(data, sizeof(BYTE), size, file);
    
    // Convert the pixel values to float
    bitmap.data = (float *)malloc(sizeof(float) * size);
    
    for (int i = 0; i < size; i++)
    {
        bitmap.data[i] = (float)data[i];
    }
    
    fclose(file);
    free(data);
    
    return bitmap;
}

/**
 * Writes a BMP file in grayscale given its image data and a filename.
 */
void writeBMPGrayscale(int width, int height, float *image, char *filename)
{
    FILE *file = NULL;
    
    file = fopen(filename, "wb");
    
    // Write the reference header
    fwrite(g_info, sizeof(BYTE), HEADER_SIZE, file);
    
    // Unwrap the 8-bit grayscale into a 24bpp (for simplicity)
    for (int h = 0; h < height; h++)
    {
        int offset = h * width;
        
        for (int w = 0; w < width; w++)
        {
            BYTE pixel = (BYTE)((image[offset + w] > 255.0f) ? 255.0f :
                                (image[offset + w] < 0.0f)   ? 0.0f   :
                                                               image[offset + w]);
            
            // Repeat the same pixel value for BGR
            fputc(pixel, file);
            fputc(pixel, file);
            fputc(pixel, file);
        }
    }
    
    fclose(file);
}

/**
 * Releases a given BMPImage.
 */
void freeBMP(BMPImage bitmap)
{
    free(bitmap.data);
}

/**
 * Checks if there has been any CUDA error. The method will automatically print
 * some information and exit the program when an error is found.
 */
void checkCUDAError()
{
    hipError_t hipError_t = hipGetLastError();
    
    if(hipError_t != hipSuccess)
    {
        printf("CUDA Error: Returned %d: %s\n", hipError_t,
                                                hipGetErrorString(hipError_t));
        exit(-1);
    }
}

/**
 * Calculates the elapsed time between two time intervals (in milliseconds).
 */
double get_elapsed(tval t0, tval t1)
{
    return (double)(t1.tv_sec - t0.tv_sec) * 1000.0L + (double)(t1.tv_usec - t0.tv_usec) / 1000.0L;
}

/**
 * Stores the result image and prints a message.
 */
void store_result(int index, double elapsed_cpu, double elapsed_gpu,
                     int width, int height, float *image)
{
    char path[255];
    
    sprintf(path, "images/hw3_result_%d.bmp", index);
    writeBMPGrayscale(width, height, image, path);
    
    printf("Step #%d Completed - Result stored in \"%s\".\n", index, path);
    printf("Elapsed CPU: %fms / ", elapsed_cpu);
    
    if (elapsed_gpu == 0)
    {
        printf("[GPU version not available]\n");
    }
    else
    {
        printf("Elapsed GPU: %fms\n", elapsed_gpu);
    }
}

/**
 * Converts a given 24bpp image into 8bpp grayscale using the CPU.
 */
void cpu_grayscale(int width, int height, float *image, float *image_out)
{
    for (int h = 0; h < height; h++)
    {
        int offset_out = h * width;      // 1 color per pixel
        int offset     = offset_out * 3; // 3 colors per pixel
        
        for (int w = 0; w < width; w++)
        {
            float *pixel = &image[offset + w * 3];
            
            // Convert to grayscale following the "luminance" model
            image_out[offset_out + w] = pixel[0] * 0.0722f + // B
                                        pixel[1] * 0.7152f + // G
                                        pixel[2] * 0.2126f;  // R
        }
    }
}

/**
 * Converts a given 24bpp image into 8bpp grayscale using the GPU.
 */
__global__ void gpu_grayscale(int width, int height, float *image, float *image_out)
{
/*             w
        -----------------    
        |BGR|BGR|BGR ...
      h |..
        |.
*/

    auto idw = blockIdx.x * blockDim.x + threadIdx.x;
    auto idh = blockIdx.y * blockDim.y + threadIdx.y;
    // Size: amount of pixels per thread.
    auto w_size = width / (blockDim.x * gridDim.x);
    auto h_size = height / (blockDim.y * gridDim.y);
    
    if (w_size == 0) w_size = 1;
    if (h_size == 0) h_size = 1;

    for (int h = idh * h_size; h < (idh+1) * h_size && h < height; h++) {
        int offset_out = h * width;      // 1 color per pixel
        int offset     = offset_out * 3; // 3 colors per pixel

        for (int w = idw * w_size; w < (idw+1) * w_size && w < width; w++) {

            float *pixel = &image[offset + w * 3];
            // Convert to grayscale following the "luminance" model
            image_out[offset_out + w] = pixel[0] * 0.0722f + // B
                                        pixel[1] * 0.7152f + // G
                                        pixel[2] * 0.2126f;  // R
        }
    }

}

/**
 * Applies a 3x3 convolution matrix to a pixel using the CPU.
 */
float cpu_applyFilter(float *image, int stride, float *matrix, int filter_dim)
{
    float pixel = 0.0f;
    
    for (int h = 0; h < filter_dim; h++)
    {
        int offset        = h * stride;
        int offset_kernel = h * filter_dim;
        
        for (int w = 0; w < filter_dim; w++)
        {
            pixel += image[offset + w] * matrix[offset_kernel + w];
        }
    }
    
    return pixel;
}

/**
 * Applies a 3x3 convolution matrix to a pixel using the GPU.
 */
__device__ float gpu_applyFilter(float *image, int stride, float *matrix, int filter_dim)
{
    ////////////////
    // TO-DO #5.2 ////////////////////////////////////////////////
    // Implement the GPU version of cpu_applyFilter()           //
    //                                                          //
    // Does it make sense to have a separate gpu_applyFilter()? //
    //
    // > No it makes no sense as both function perform the same
    // things. So one should code only one and mark it with
    // __host__ __device__ so it can run on both cpu and gpu.
    //
    // > even the changes brung by the shared memory part implied
    // no change to this perfect function.
    //////////////////////////////////////////////////////////////
    
    float pixel = 0.0f;

    for (int h = 0; h < filter_dim; h++)
    {
        int offset        = h * stride;
        int offset_kernel = h * filter_dim;

        for (int w = 0; w < filter_dim; w++)
        {
            pixel += image[offset + w] * matrix[offset_kernel + w];
        }
    }

    return pixel;
}

/**
 * Applies a Gaussian 3x3 filter to a given image using the CPU.
 */
void cpu_gaussian(int width, int height, float *image, float *image_out)
{
    float gaussian[9] = { 1.0f / 16.0f, 2.0f / 16.0f, 1.0f / 16.0f,
                          2.0f / 16.0f, 4.0f / 16.0f, 2.0f / 16.0f,
                          1.0f / 16.0f, 2.0f / 16.0f, 1.0f / 16.0f };
    
    for (int h = 0; h < (height - 2); h++)
    {
        int offset_t = h * width;
        int offset   = (h + 1) * width;
        
        for (int w = 0; w < (width - 2); w++)
        {
            image_out[offset + (w + 1)] = cpu_applyFilter(&image[offset_t + w],
                                                          width, gaussian, 3);
        }
    }
}

/**
 * Applies a Gaussian 3x3 filter to a given image using the GPU.
 */
__global__ void gpu_gaussian(int width, int height, float *image, float *image_out)
{

    __shared__ float sh_block[BLOCK_SIZE_SH * BLOCK_SIZE_SH];

    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Copy ONE pixel from the input image to the shared block
    sh_block[(threadIdx.y + 1)* BLOCK_SIZE_SH + threadIdx.x + 1] = image[index_y * width + index_x];

    // take care of the last row or first row or first column or last column
    if (threadIdx.x == 0 && index_x != 0) {

        sh_block[(threadIdx.y + 1)* BLOCK_SIZE_SH]
                                  = image[index_y * width + index_x - 1];

        if (threadIdx.y == 0 && index_y != 0)
            sh_block[0] = image[(index_y - 1) * width + index_x - 1];

    } else if (threadIdx.x == blockDim.x - 1 && index_x != width - 1) {
         
         
        sh_block[(threadIdx.y + 1)* BLOCK_SIZE_SH + threadIdx.x + 2]
                                  = image[index_y * width + index_x + 1];

        if (threadIdx.y == blockDim.y - 1 && index_y != height - 1)
            sh_block[BLOCK_SIZE_SH*BLOCK_SIZE_SH-1] = image[(index_y + 1) * width + index_x + 1];
         
    } else if (threadIdx.y == 0 && index_y != 0) {

        sh_block[threadIdx.x + 1]
                                  = image[(index_y - 1) * width + index_x];
        
        if (threadIdx.x == blockDim.x - 1 && index_x != width - 1)
            sh_block[threadIdx.x + 2] = image[(index_y - 1) * width + index_x + 1];

    } else if (threadIdx.y == blockDim.y - 1 && index_y != height - 1) {
         
         
        sh_block[(threadIdx.y + 2) * BLOCK_SIZE_SH + threadIdx.x + 1]
                                  = image[(index_y + 1) * width + index_x];

        if (threadIdx.x == 0 && index_x != 0)
            sh_block[(threadIdx.y+2) * BLOCK_SIZE_SH] = image[(index_y + 1) * width + index_x - 1];
         
    }

    __syncthreads();

    static float gaussian[9] = { 1.0f / 16.0f, 2.0f / 16.0f, 1.0f / 16.0f,
                          2.0f / 16.0f, 4.0f / 16.0f, 2.0f / 16.0f,
                          1.0f / 16.0f, 2.0f / 16.0f, 1.0f / 16.0f };
    
    if (index_x < (width - 2) && index_y < (height - 2))
    {
        int offset_sh = threadIdx.y * BLOCK_SIZE_SH + threadIdx.x;
        int offset = (index_y + 1) * width + (index_x + 1);
        
        image_out[offset] = gpu_applyFilter(&sh_block[offset_sh],
                                       BLOCK_SIZE_SH, gaussian, 3);
    }
}

/**
 * Calculates the gradient of an image using a Sobel filter on the CPU.
 */
void cpu_sobel(int width, int height, float *image, float *image_out)
{
    float sobel_x[9] = { 1.0f,  0.0f, -1.0f,
                         2.0f,  0.0f, -2.0f,
                         1.0f,  0.0f, -1.0f };
    float sobel_y[9] = { 1.0f,  2.0f,  1.0f,
                         0.0f,  0.0f,  0.0f,
                        -1.0f, -2.0f, -1.0f };
    
    for (int h = 0; h < (height - 2); h++)
    {
        int offset_t = h * width;
        int offset   = (h + 1) * width;
        
        for (int w = 0; w < (width - 2); w++)
        {
            float gx = cpu_applyFilter(&image[offset_t + w], width, sobel_x, 3);
            float gy = cpu_applyFilter(&image[offset_t + w], width, sobel_y, 3);
            
            // Note: The output can be negative or exceed the max. color value
            // of 255. We compensate this afterwards while storing the file.
            image_out[offset + (w + 1)] = sqrtf(gx * gx + gy * gy);
        }
    }
}

/**
 * Calculates the gradient of an image using a Sobel filter on the GPU.
 */
__global__ void gpu_sobel(int width, int height, float *image, float *image_out)
{
    ////////////////
    // TO-DO #6.1 /////////////////////////////////////
    // Implement the GPU version of the Sobel filter //
    // It ressembles the gpu_gaussian. Somehow.      //
    ///////////////////////////////////////////////////

    __shared__ float sh_block[BLOCK_SIZE_SH * BLOCK_SIZE_SH];

    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;

    // copy one own pixel.
    sh_block[(threadIdx.y + 1)* BLOCK_SIZE_SH + threadIdx.x + 1] = image[index_y * width + index_x];

    // take care of the last row or first row or first column or last column
    if (threadIdx.x == 0 && index_x != 0) {

        sh_block[(threadIdx.y + 1)* BLOCK_SIZE_SH]
                                  = image[index_y * width + index_x - 1];

        if (threadIdx.y == 0 && index_y != 0)
            sh_block[0] = image[(index_y - 1) * width + index_x - 1];

    } else if (threadIdx.x == blockDim.x - 1 && index_x != width - 1) {
         
         
        sh_block[(threadIdx.y + 1)* BLOCK_SIZE_SH + threadIdx.x + 2]
                                  = image[index_y * width + index_x + 1];

        if (threadIdx.y == blockDim.y - 1 && index_y != height - 1)
            sh_block[BLOCK_SIZE_SH*BLOCK_SIZE_SH-1] = image[(index_y + 1) * width + index_x + 1];
         
    } else if (threadIdx.y == 0 && index_y != 0) {

        sh_block[threadIdx.x + 1]
                                  = image[(index_y - 1) * width + index_x];
        
        if (threadIdx.x == blockDim.x - 1 && index_x != width - 1)
            sh_block[threadIdx.x + 2] = image[(index_y - 1) * width + index_x + 1];

    } else if (threadIdx.y == blockDim.y - 1 && index_y != height - 1) {
         
         
        sh_block[(threadIdx.y + 2) * BLOCK_SIZE_SH + threadIdx.x + 1]
                                  = image[(index_y + 1) * width + index_x];

        if (threadIdx.x == 0 && index_x != 0)
            sh_block[(threadIdx.y+2) * BLOCK_SIZE_SH] = image[(index_y + 1) * width + index_x - 1];
         
    }

    __syncthreads();

    static float sobel_x[9] = { 1.0f,  0.0f, -1.0f,
                         2.0f,  0.0f, -2.0f,
                         1.0f,  0.0f, -1.0f };
    static float sobel_y[9] = { 1.0f,  2.0f,  1.0f,
                         0.0f,  0.0f,  0.0f,
                        -1.0f, -2.0f, -1.0f };

    if (index_y < height - 2 && index_x < width - 2) {

        int offset_sh = (threadIdx.y * BLOCK_SIZE_SH) + threadIdx.x;

        float gx = gpu_applyFilter(&sh_block[offset_sh], BLOCK_SIZE_SH, sobel_x, 3);
        float gy = gpu_applyFilter(&sh_block[offset_sh], BLOCK_SIZE_SH, sobel_y, 3);

        image_out[((index_y+1) * width) + (index_x + 1)] = sqrtf(gx*gx + gy*gy);
    }

/*    auto idw = blockIdx.x * blockDim.x + threadIdx.x;
    auto idh = blockIdx.y * blockDim.y + threadIdx.y;
    // Size: amount of pixels per thread.
    auto w_size = (width-2) / (blockDim.x * gridDim.x);
    auto h_size = (height-2) / (blockDim.y * gridDim.y);

    if (w_size == 0) w_size = 1;
    if (h_size == 0) h_size = 1;

    for (int h = idh * h_size; h < (idh+1) * h_size && h < height - 2; h++) {
        int offset_in = h * width;
        int offset    = (h+1) * width; // Ignore (first & last)-(row & column) of image_out.

        for (int w = idw * w_size; w < (idw+1) * w_size && w < width - 2; w++) {

            float gx = gpu_applyFilter(&image[offset_in + w], width, sobel_x, 3);
            float gy = gpu_applyFilter(&image[offset_in + w], width, sobel_y, 3);

            image_out[offset + (w + 1)] = sqrtf(gx*gx + gy*gy);
        }
    }
*/
}

int main(int argc, char **argv)
{
    BMPImage bitmap          = { 0 };
    float    *d_bitmap       = { 0 };
    float    *image_out[2]   = { 0 };
    float    *d_image_out[2] = { 0 };
    int      image_size      = 0;
    tval     t[2]            = { 0 };
    double   elapsed[2]      = { 0 };
    dim3     grid(1);                       // The grid will be defined later
    dim3     block(BLOCK_SIZE, BLOCK_SIZE); // The block size will not change
    
    double   total_cpu_time = 0.0L;
    double   total_gpu_time = 0.0L;

    // Make sure the filename is provided
    if (argc != 2)
    {
        fprintf(stderr, "Error: The filename is missing!\n");
        return -1;
    }
    
    // Read the input image and update the grid dimension
    bitmap     = readBMP(argv[1]);
    image_size = bitmap.width * bitmap.height;
    grid       = dim3(((bitmap.width  + (BLOCK_SIZE - 1)) / BLOCK_SIZE),
                      ((bitmap.height + (BLOCK_SIZE - 1)) / BLOCK_SIZE));
    
    printf("Image opened (width=%d height=%d).\n", bitmap.width, bitmap.height);
    printf("Let's do this with %dx%d blocks with %dx%d threads each.\n",
                                            grid.x, grid.y, block.x, block.y);
 
    // Allocate the intermediate image buffers for each step
    for (int i = 0; i < 2; i++)
    {
        image_out[i] = (float *)calloc(image_size, sizeof(float));
        
        hipMalloc(&d_image_out[i], image_size * sizeof(float));
        hipMemset(d_image_out[i], 0, image_size * sizeof(float));
    }

    hipMalloc(&d_bitmap, image_size * sizeof(float) * 3);
    hipMemcpy(d_bitmap, bitmap.data,
               image_size * sizeof(float) * 3, hipMemcpyHostToDevice);
    
    // Step 1: Convert to grayscale
    {
        // Launch the CPU version
        gettimeofday(&t[0], NULL);
        //cpu_grayscale(bitmap.width, bitmap.height, bitmap.data, image_out[0]);
        gettimeofday(&t[1], NULL);
        
        elapsed[0] = get_elapsed(t[0], t[1]);
        
        // Launch the GPU version
        gettimeofday(&t[0], NULL);
        gpu_grayscale<<<grid, block>>>(bitmap.width, bitmap.height,
                                       d_bitmap, d_image_out[0]);
        
	hipMemcpy(image_out[0], d_image_out[0],
                   image_size * sizeof(float), hipMemcpyDeviceToHost);
        gettimeofday(&t[1], NULL);
        
        elapsed[1] = get_elapsed(t[0], t[1]);
        
        // Store the result image in grayscale
        store_result(1, elapsed[0], elapsed[1], bitmap.width, bitmap.height, image_out[0]);
        total_cpu_time += elapsed[0];
        total_gpu_time += elapsed[1];
    }
    
    // Step 2: Apply a 3x3 Gaussian filter
    {
        // Launch the CPU version
        gettimeofday(&t[0], NULL);
        //cpu_gaussian(bitmap.width, bitmap.height, image_out[0], image_out[1]);
        gettimeofday(&t[1], NULL);
        
        elapsed[0] = get_elapsed(t[0], t[1]);
        
        // Launch the GPU version
        gettimeofday(&t[0], NULL);
        gpu_gaussian<<<grid, block>>>(bitmap.width, bitmap.height,
                                      d_image_out[0], d_image_out[1]);
        
        hipMemcpy(image_out[1], d_image_out[1],
                   image_size * sizeof(float), hipMemcpyDeviceToHost);
        gettimeofday(&t[1], NULL);
        
        elapsed[1] = get_elapsed(t[0], t[1]);
        
        // Store the result image with the Gaussian filter applied
        store_result(2, elapsed[0], elapsed[1], bitmap.width, bitmap.height, image_out[1]);
       	total_cpu_time += elapsed[0];
       	total_gpu_time += elapsed[1];
    }
    
    // Step 3: Apply a Sobel filter
    {
        // Launch the CPU version
        gettimeofday(&t[0], NULL);
        //cpu_sobel(bitmap.width, bitmap.height, image_out[1], image_out[0]);
        gettimeofday(&t[1], NULL);
        
        elapsed[0] = get_elapsed(t[0], t[1]);
        
        // Launch the GPU version
        gettimeofday(&t[0], NULL);
        gpu_sobel<<<grid, block>>>(bitmap.width, bitmap.height,
                                   d_image_out[1], d_image_out[0]);
        
        hipMemcpy(image_out[0], d_image_out[0],
                   image_size * sizeof(float), hipMemcpyDeviceToHost);
        gettimeofday(&t[1], NULL);
        
        elapsed[1] = get_elapsed(t[0], t[1]);
        
        // Store the final result image with the Sobel filter applied
        store_result(3, elapsed[0], elapsed[1], bitmap.width, bitmap.height, image_out[0]);
       	total_cpu_time += elapsed[0];
       	total_gpu_time += elapsed[1];
    }
    
    // Release the allocated memory
    for (int i = 0; i < 2; i++)
    {
        free(image_out[i]);
        hipFree(d_image_out[i]);
    }
    
    freeBMP(bitmap);
    hipFree(d_bitmap);

    printf("Total CPU time: %lf, total GPU time: %lf\n", total_cpu_time, total_gpu_time);
    
    return 0;
}
