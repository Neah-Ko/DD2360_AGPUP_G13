#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define TPB 256
#define ARRAY_SIZE 1000000
#define BOUND_RAND 100
#define FLOAT_TH 1e-2

__global__ void saxpyKernel(size_t n, const float a, const float * d_x, float * d_y){
	/* get index */
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i > n) return;
	/* compute */
	d_y[i] += a * d_x[i];
}

void saxpyLauncher(size_t n, const float a, const float * x, float * y){
	/* Copies input arrays on the GPU and calls saxpy kernel */
	float * d_x;
	float * d_y;

	hipMalloc(&d_x, n*sizeof(float));
	hipMalloc(&d_y, n*sizeof(float));

	hipMemcpy(d_x, x, n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, n*sizeof(float), hipMemcpyHostToDevice);

	saxpyKernel<<<(n + TPB - 1)/TPB, TPB>>>(n, a, d_x, d_y);
	hipMemcpy(y, d_y, n*sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_x);
	hipFree(d_y);
}


void saxpy(size_t n, const float a, const float * x, float * y){
	for(size_t i = 0; i < n; i++){
		y[i] += a*x[i];
	}
}

float randFloat(){
	return ((float)(rand() / (float)RAND_MAX)) * 2 * BOUND_RAND - BOUND_RAND;
}

double cpuSecond() {
   struct timeval tp;
   gettimeofday(&tp,NULL);
   return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}


int main(){
	/* variables */
	float x[ARRAY_SIZE];
	float y[ARRAY_SIZE];
	float y_k[ARRAY_SIZE];
	float a;
	printf("ARRAY_SIZE = %d\n", ARRAY_SIZE);

	/* generate data */
	srand(time(NULL)); // seed
	a = randFloat();
	for(size_t i = 0; i < ARRAY_SIZE; i++){
		x[i] = randFloat(); 
		y[i] = randFloat();
		y_k[i] = y[i];
	}

	/* CPU version */
	printf("Computing SAXPY on the CPU… ");
	double cpu_iStart = cpuSecond();
	saxpy(ARRAY_SIZE, a, x, y);
	double cpu_iElaps = cpuSecond() - cpu_iStart;
	printf("Done! in %f seconds\n", cpu_iElaps);


	/* GPU version */
	printf("Computing SAXPY on the GPU… ");
	double gpu_iStart = cpuSecond();
	saxpyLauncher(ARRAY_SIZE, a, x, y_k);
	hipDeviceSynchronize();
	double gpu_iElaps = cpuSecond() - gpu_iStart;
	printf("Done! in %f seconds\n", gpu_iElaps);

	/* Compare */
	printf("Comparing the output for each implementation… ");
	for(size_t i = 0; i < ARRAY_SIZE; i++){
		if((y[i] - y_k[i]) > FLOAT_TH) {
			printf("Incorrect!\n");
			return -1;
		}
	}
	printf("Correct!\n");
	return 0;
}