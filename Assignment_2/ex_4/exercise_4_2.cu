
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>

// #define NUM_ITER 1000000000
#define NUM_ITER 10000000
#define NUM_ITER_KERNEL 100
#define TPB 128


double cpuSecond() {
   struct timeval tp;
   gettimeofday(&tp,NULL);
   return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

__global__ void piKernel(size_t n, int * d_counts, hiprandState * states){
	/* get index */
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i > n) return;

	double x;
	double y;
	double z;

    hiprand_init(i, i, 0, &states[i]);

    for(int j = 0; j < NUM_ITER_KERNEL; j++){
    	x = hiprand_uniform(&states[i]);
    	y = hiprand_uniform(&states[i]);
    	z = sqrt((x*x) + (y*y));
    	if (z <= 1.0) {
            d_counts[i]++;
        }
    }
}

void piLauncher(size_t n, int * counts){
	int * d_counts;
	hiprandState *dev_random;


	hipMalloc((void**)&dev_random, n*sizeof(hiprandState));
	hipMalloc(&d_counts, n*sizeof(int));
	hipMemset(d_counts, 0, n*sizeof(int));

	piKernel<<<(n + TPB -1), TPB>>>(n, d_counts, dev_random);

	hipMemcpy(counts, d_counts, n*sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_random);
	hipFree(d_counts);
}


int main(int argc, char **argv){
	/* variables */
	int n = NUM_ITER / NUM_ITER_KERNEL;
	int counts[n];
	int count = 0;
	double pi;

	/* Compute pi on the GPU */
	printf("Estimating pi on the GPU… ");
	double gpu_iStart = cpuSecond();
	piLauncher(n, counts);
	hipDeviceSynchronize();

	for(int i = 0; i < n; i++){
		count += counts[i];
	}
	pi = ((double)count / (double)NUM_ITER) * 4.0;

	double gpu_iElaps = cpuSecond() - gpu_iStart;
	printf("Done! in %f seconds\n", gpu_iElaps);

	printf("The result is %f\n", pi);

	return 0;
}